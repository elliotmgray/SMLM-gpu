#include "hip/hip_runtime.h"

#include "stdio.h"
int main()
{
    double *matrix, *d_A;
    
    matrix = (double *)calloc(1000, sizeof(double));
    hipMalloc( &d_A, 1000*sizeof(double));
    hipMemcpy(d_A, matrix, 1000*sizeof(double), hipMemcpyHostToDevice);
    
    printf("\nthe first element of matrix is %f\n", matrix[0]);
    
    hipFree(d_A);
    free(matrix);
    return 0;
}